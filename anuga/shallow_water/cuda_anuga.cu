
#include <hip/hip_runtime.h>
// CUDA code to implement flux and quantity updates as used 
// in evolve loop



// FIXME SR: this routine doesn't seem to be used in flux calculation, probably used in 
// extrapolation
__device__ int __find_qmin_and_qmax(double dq0, double dq1, double dq2,
  double *qmin, double *qmax)
{
// Considering the centroid of an FV triangle and the vertices of its
// auxiliary triangle, find
// qmin=min(q)-qc and qmax=max(q)-qc,
// where min(q) and max(q) are respectively min and max over the
// four values (at the centroid of the FV triangle and the auxiliary
// triangle vertices),
// and qc is the centroid
// dq0=q(vertex0)-q(centroid of FV triangle)
// dq1=q(vertex1)-q(vertex0)
// dq2=q(vertex2)-q(vertex0)

// This is a simple implementation
*qmax = fmax(fmax(dq0, fmax(dq0 + dq1, dq0 + dq2)), 0.0);
*qmin = fmin(fmin(dq0, fmin(dq0 + dq1, dq0 + dq2)), 0.0);

return 0;
}

// FIXME SR: this routine doesn't seem to be used in flux calculation, probably used in 
// extrapolation
__device__ int __limit_gradient(double *dqv, double qmin, double qmax, double beta_w)
{
// Given provisional jumps dqv from the FV triangle centroid to its
// vertices/edges, and jumps qmin (qmax) between the centroid of the FV
// triangle and the minimum (maximum) of the values at the auxiliary triangle
// vertices (which are centroids of neighbour mesh triangles), calculate a
// multiplicative factor phi by which the provisional vertex jumps are to be
// limited

int i;
double r = 1000.0, r0 = 1.0, phi = 1.0;
static double TINY = 1.0e-100; // to avoid machine accuracy problems.
// FIXME: Perhaps use the epsilon used elsewhere.

// Any provisional jump with magnitude < TINY does not contribute to
// the limiting process.
// return 0;

for (i = 0; i < 3; i++)
{
if (dqv[i] < -TINY)
r0 = qmin / dqv[i];

if (dqv[i] > TINY)
r0 = qmax / dqv[i];

r = fmin(r0, r);
}

phi = fmin(r * beta_w, 1.0);
// phi=1.;
dqv[0] = dqv[0] * phi;
dqv[1] = dqv[1] * phi;
dqv[2] = dqv[2] * phi;

return 0;
}

// Computational function for rotation
__device__ void __rotate(double *q, double n1, double n2)
{
  /*Rotate the last  2 coordinates of q (q[1], q[2])
    from x,y coordinates to coordinates based on normal vector (n1, n2).

    Result is returned in array 2x1 r
    To rotate in opposite direction, call rotate with (q, n1, -n2)

    Contents of q are changed by this function */

  double q1, q2;

  // Shorthands
  q1 = q[1]; // x coordinate
  q2 = q[2]; // y coordinate

  // Rotate
  q[1] = n1 * q1 + n2 * q2;
  q[2] = -n2 * q1 + n1 * q2;

}

// Innermost flux function (using stage w=z+h)
__device__ void __flux_function_central(double *q_left, double *q_right,
                            double h_left, double h_right,
                            double hle, double hre,
                            double n1, double n2,
                            double epsilon,
                            double ze,
                            double limiting_threshold,
                            double g,
                            double *edgeflux, double *max_speed,
                            double *pressure_flux, double hc,
                            double hc_n,
                            long low_froude)
{

  /*Compute fluxes between volumes for the shallow water wave equation
    cast in terms of the 'stage', w = h+z using
    the 'central scheme' as described in

    Kurganov, Noelle, Petrova. 'Semidiscrete Central-Upwind Schemes For
    Hyperbolic Conservation Laws and Hamilton-Jacobi Equations'.
    Siam J. Sci. Comput. Vol. 23, No. 3, pp. 707-740.

    The implemented formula is given in equation (3.15) on page 714

    FIXME: Several variables in this interface are no longer used, clean up
  */

  int i;

  double uh_left, vh_left, u_left;
  double uh_right, vh_right, u_right;
  double s_min, s_max, soundspeed_left, soundspeed_right;
  double denom, inverse_denominator;
  double tmp, local_fr, v_right, v_left;
  double q_left_rotated[3], q_right_rotated[3], flux_right[3], flux_left[3];

  if (h_left == 0. && h_right == 0.)
  {
    // Quick exit
    memset(edgeflux, 0, 3 * sizeof(double));
    *max_speed = 0.0;
    *pressure_flux = 0.;
    return;
  }
  // Copy conserved quantities to protect from modification
  q_left_rotated[0] = q_left[0];
  q_right_rotated[0] = q_right[0];
  q_left_rotated[1] = q_left[1];
  q_right_rotated[1] = q_right[1];
  q_left_rotated[2] = q_left[2];
  q_right_rotated[2] = q_right[2];

  // Align x- and y-momentum with x-axis
  __rotate(q_left_rotated, n1, n2);
  __rotate(q_right_rotated, n1, n2);

  // Compute speeds in x-direction
  // w_left = q_left_rotated[0];
  uh_left = q_left_rotated[1];
  vh_left = q_left_rotated[2];
  if (hle > 0.0)
  {
    tmp = 1.0 / hle;
    u_left = uh_left * tmp; // max(h_left, 1.0e-06);
    uh_left = h_left * u_left;
    v_left = vh_left * tmp; // Only used to define local_fr
    vh_left = h_left * tmp * vh_left;
  }
  else
  {
    u_left = 0.;
    uh_left = 0.;
    vh_left = 0.;
    v_left = 0.;
  }

  // u_left = _compute_speed(&uh_left, &hle,
  //             epsilon, h0, limiting_threshold);

  // w_right = q_right_rotated[0];
  uh_right = q_right_rotated[1];
  vh_right = q_right_rotated[2];
  if (hre > 0.0)
  {
    tmp = 1.0 / hre;
    u_right = uh_right * tmp; // max(h_right, 1.0e-06);
    uh_right = h_right * u_right;
    v_right = vh_right * tmp; // Only used to define local_fr
    vh_right = h_right * tmp * vh_right;
  }
  else
  {
    u_right = 0.;
    uh_right = 0.;
    vh_right = 0.;
    v_right = 0.;
  }
  // u_right = _compute_speed(&uh_right, &hre,
  //               epsilon, h0, limiting_threshold);

  // Maximal and minimal wave speeds
  soundspeed_left = sqrt(g * h_left);
  soundspeed_right = sqrt(g * h_right);
  // soundspeed_left  = sqrt(g*hle);
  // soundspeed_right = sqrt(g*hre);

  // Something that scales like the Froude number
  // We will use this to scale the diffusive component of the UH/VH fluxes.

  // low_froude can have values 0, 1, 2
  if (low_froude == 1)
  {
    local_fr = sqrt(
        fmax(0.001, fmin(1.0,
                         (u_right * u_right + u_left * u_left + v_right * v_right + v_left * v_left) /
                             (soundspeed_left * soundspeed_left + soundspeed_right * soundspeed_right + 1.0e-10))));
  }
  else if (low_froude == 2)
  {
    local_fr = sqrt((u_right * u_right + u_left * u_left + v_right * v_right + v_left * v_left) /
                    (soundspeed_left * soundspeed_left + soundspeed_right * soundspeed_right + 1.0e-10));
    local_fr = sqrt(fmin(1.0, 0.01 + fmax(local_fr - 0.01, 0.0)));
  }
  else
  {
    local_fr = 1.0;
  }
  // printf("local_fr %e \n:", local_fr);

  s_max = fmax(u_left + soundspeed_left, u_right + soundspeed_right);
  if (s_max < 0.0)
  {
    s_max = 0.0;
  }

  // if( hc < 1.0e-03){
  //   s_max = 0.0;
  // }

  s_min = fmin(u_left - soundspeed_left, u_right - soundspeed_right);
  if (s_min > 0.0)
  {
    s_min = 0.0;
  }

  // if( hc_n < 1.0e-03){
  //   s_min = 0.0;
  // }

  // Flux formulas
  flux_left[0] = u_left * h_left;
  flux_left[1] = u_left * uh_left; //+ 0.5*g*h_left*h_left;
  flux_left[2] = u_left * vh_left;

  flux_right[0] = u_right * h_right;
  flux_right[1] = u_right * uh_right; //+ 0.5*g*h_right*h_right;
  flux_right[2] = u_right * vh_right;

  // Flux computation
  denom = s_max - s_min;
  if (denom < epsilon)
  {
    // Both wave speeds are very small
    //memset(edgeflux, 0, 3 * sizeof(double)); 
    edgeflux[0] = 0.0;
    edgeflux[1] = 0.0;
    edgeflux[2] = 0.0;


    *max_speed = 0.0;
    //*pressure_flux = 0.0;
    *pressure_flux = 0.5 * g * 0.5 * (h_left * h_left + h_right * h_right);
  }
  else
  {
    // Maximal wavespeed
    *max_speed = fmax(s_max, -s_min);

    inverse_denominator = 1.0 / fmax(denom, 1.0e-100);
    for (i = 0; i < 3; i++)
    {
      edgeflux[i] = s_max * flux_left[i] - s_min * flux_right[i];

      // Standard smoothing term
      // edgeflux[i] += 1.0*(s_max*s_min)*(q_right_rotated[i] - q_left_rotated[i]);
      // Smoothing by stage alone can cause high velocities / slow draining for nearly dry cells
      if (i == 0)
        edgeflux[i] += (s_max * s_min) * (fmax(q_right_rotated[i], ze) - fmax(q_left_rotated[i], ze));
      // if(i==0) edgeflux[i] += (s_max*s_min)*(h_right - h_left);
      if (i == 1)
        edgeflux[i] += local_fr * (s_max * s_min) * (uh_right - uh_left);
      if (i == 2)
        edgeflux[i] += local_fr * (s_max * s_min) * (vh_right - vh_left);

      edgeflux[i] *= inverse_denominator;
    }
    // Separate pressure flux, so we can apply different wet-dry hacks to it
    *pressure_flux = 0.5 * g * (s_max * h_left * h_left - s_min * h_right * h_right) * inverse_denominator;

    // Rotate back
    __rotate(edgeflux, n1, -n2);
  }

}


__device__ double __adjust_edgeflux_with_weir(double *edgeflux,
                                   double h_left, double h_right,
                                   double g, double weir_height,
                                   double Qfactor,
                                   double s1, double s2,
                                   double h1, double h2,
                                   double *max_speed_local)
{
  // Adjust the edgeflux to agree with a weir relation [including
  // subergence], but smoothly vary to shallow water solution when
  // the flow over the weir is much deeper than the weir, or the
  // upstream/downstream water elevations are too similar
  double rw, rw2; // 'Raw' weir fluxes
  double rwRat, hdRat, hdWrRat, scaleFlux, minhd, maxhd;
  double w1, w2; // Weights for averaging
  double newFlux;
  double twothirds = (2.0 / 3.0);
  // Following constants control the 'blending' with the shallow water solution
  // They are now user-defined
  // double s1=0.9; // At this submergence ratio, begin blending with shallow water solution
  // double s2=0.95; // At this submergence ratio, completely use shallow water solution
  // double h1=1.0; // At this (tailwater height above weir) / (weir height) ratio, begin blending with shallow water solution
  // double h2=1.5; // At this (tailwater height above weir) / (weir height) ratio, completely use the shallow water solution

  if ((h_left <= 0.0) && (h_right <= 0.0))
  {
    return 0;
  }

  minhd = fmin(h_left, h_right);
  maxhd = fmax(h_left, h_right);
  // 'Raw' weir discharge = Qfactor*2/3*H*(2/3*g*H)**0.5
  rw = Qfactor * twothirds * maxhd * sqrt(twothirds * g * maxhd);
  // Factor for villemonte correction
  rw2 = Qfactor * twothirds * minhd * sqrt(twothirds * g * minhd);
  // Useful ratios
  rwRat = rw2 / fmax(rw, 1.0e-100);
  hdRat = minhd / fmax(maxhd, 1.0e-100);

  // (tailwater height above weir)/weir_height ratio
  hdWrRat = minhd / fmax(weir_height, 1.0e-100);

  // Villemonte (1947) corrected weir flow with submergence
  // Q = Q1*(1-Q2/Q1)**0.385
  rw = rw * pow(1.0 - rwRat, 0.385);

  if (h_right > h_left)
  {
    rw *= -1.0;
  }

  if ((hdRat < s2) & (hdWrRat < h2))
  {
    // Rescale the edge fluxes so that the mass flux = desired flux
    // Linearly shift to shallow water solution between hdRat = s1 and s2
    // and between hdWrRat = h1 and h2

    //
    // WEIGHT WITH RAW SHALLOW WATER FLUX BELOW
    // This ensures that as the weir gets very submerged, the
    // standard shallow water equations smoothly take over
    //

    // Weighted average constants to transition to shallow water eqn flow
    w1 = fmin(fmax(hdRat - s1, 0.) / (s2 - s1), 1.0);

    // Adjust again when the head is too deep relative to the weir height
    w2 = fmin(fmax(hdWrRat - h1, 0.) / (h2 - h1), 1.0);

    newFlux = (rw * (1.0 - w1) + w1 * edgeflux[0]) * (1.0 - w2) + w2 * edgeflux[0];

    if (fabs(edgeflux[0]) > 1.0e-100)
    {
      scaleFlux = newFlux / edgeflux[0];
    }
    else
    {
      scaleFlux = 0.;
    }

    scaleFlux = fmax(scaleFlux, 0.);

    edgeflux[0] = newFlux;

    // FIXME: Do this in a cleaner way
    // IDEA: Compute momentum flux implied by weir relations, and use
    //       those in a weighted average (rather than the rescaling trick here)
    // If we allow the scaling to momentum to be unbounded,
    // velocity spikes can arise for very-shallow-flooded walls
    edgeflux[1] *= fmin(scaleFlux, 10.);
    edgeflux[2] *= fmin(scaleFlux, 10.);
  }

  // Adjust the max speed
  if (fabs(edgeflux[0]) > 0.)
  {
    *max_speed_local = sqrt(g * (maxhd + weir_height)) + fabs(edgeflux[0] / (maxhd + 1.0e-12));
  }
  //*max_speed_local += fabs(edgeflux[0])/(maxhd+1.0e-100);
  //*max_speed_local *= fmax(scaleFlux, 1.0);

  return 0;
}


// FIXME SR: At present reduction is done outside kernel
__device__ double atomicMin_double(double* address, double val)

{

	    unsigned long long int* address_as_ull = (unsigned long long int*) address;

	        unsigned long long int old = *address_as_ull, assumed;

		    do {

	                      assumed = old;
			      old = atomicCAS(address_as_ull, assumed,
							                __double_as_longlong(fmin(val, __longlong_as_double(assumed))));
					        } while (assumed != old);

		        return __longlong_as_double(old);

}
// Parallel loop in cuda_compute_fluxes
// Computational function for flux computation
// need to return local_timestep and boundary_flux_sum_substep
__global__ void _cuda_compute_fluxes_loop_1(double* timestep_k_array,  // InOut
                                    double* boundary_flux_sum_k_array, // InOut
                                    double* max_speed,               // InOut
                                    double* stage_explicit_update,   // InOut
                                    double* xmom_explicit_update,    // InOut
                                    double* ymom_explicit_update,    // InOut

                                    double* stage_centroid_values,
                                    double* stage_edge_values,
                                    double* xmom_edge_values,
                                    double* ymom_edge_values,
                                    double* bed_edge_values,
                                    double* height_edge_values,
                                    double* height_centroid_values,
                                    double* bed_centroid_values,
                                    double* stage_boundary_values,
                                    double* xmom_boundary_values,
                                    double* ymom_boundary_values,
                                    double* areas,
                                    double* normals,
                                    double* edgelengths,
                                    double* radii,
                                    long* tri_full_flag,
                                    long* neighbours,
                                    long* neighbour_edges,
                                    long* edge_flux_type,
                                    long* edge_river_wall_counter,
                                    double* riverwall_elevation,
                                    long* riverwall_rowIndex,
                                    double* riverwall_hydraulic_properties,

                                    long number_of_elements,
                                    long substep_count,
                                    long ncol_riverwall_hydraulic_properties,
                                    double epsilon,
                                    double g,
                                    long low_froude,
                                    double limiting_threshold)
{
  // #pragma omp parallel for simd default(none) shared(D, substep_count, ) \


  long k, i, ki, ki2, n, m, nm, ii;
  long RiverWall_count;
  double max_speed_local, length, inv_area, zl, zr;
  double h_left, h_right;
  double z_half, ql[3], pressuregrad_work;
  double qr[3], edgeflux[3], edge_timestep, normal_x, normal_y;
  double hle, hre, zc, zc_n, Qfactor, s1, s2, h1, h2, pressure_flux, hc, hc_n;
  double h_left_tmp, h_right_tmp, weir_height;

  // Set explicit_update to zero for all conserved_quantities.
  // This assumes compute_fluxes called before forcing terms
  double local_stage_explicit_update = 0.0;
  double local_xmom_explicit_update  = 0.0;
  double local_ymom_explicit_update  = 0.0;

  double local_max_speed = 0.0;
  double local_timestep = 1.0e+100;
  double local_boundary_flux_sum = 0.0;
  double speed_max_last = 0.0;


  //for (k = 0; k < number_of_elements; k++)
  k = blockIdx.x * blockDim.x + threadIdx.x; 
  if(k<number_of_elements)
  {

    // Loop through neighbours and compute edge flux for each
    for (i = 0; i < 3; i++)
    {
      ki = 3 * k + i; // Linear index to edge i of triangle k
      ki2 = 2 * ki;   // k*6 + i*2

      // Get left hand side values from triangle k, edge i
      ql[0] = stage_edge_values[ki];
      ql[1] = xmom_edge_values[ki];
      ql[2] = ymom_edge_values[ki];
      zl =    bed_edge_values[ki];
      hle =   height_edge_values[ki];

      hc = height_centroid_values[k];
      zc = bed_centroid_values[k];

      // Get right hand side values either from neighbouring triangle
      // or from boundary array (Quantities at neighbour on nearest face).
      n = neighbours[ki];
      hc_n = hc;
      zc_n = bed_centroid_values[k];
      if (n < 0)
      {
        // Neighbour is a boundary condition
        m = -n - 1; // Convert negative flag to boundary index

        qr[0] = stage_boundary_values[m];
        qr[1] = xmom_boundary_values[m];
        qr[2] = ymom_boundary_values[m];
        zr = zl;                     // Extend bed elevation to boundary
        hre = fmax(qr[0] - zr, 0.0); // hle;
      }
      else
      {
        // Neighbour is a real triangle
        hc_n = height_centroid_values[n];
        zc_n = bed_centroid_values[n];

        m = neighbour_edges[ki];
        nm = n * 3 + m; // Linear index (triangle n, edge m)

        qr[0] = stage_edge_values[nm];
        qr[1] = xmom_edge_values[nm];
        qr[2] = ymom_edge_values[nm];
        zr = bed_edge_values[nm];
        hre = height_edge_values[nm];
      }

      // Audusse magic for well balancing
      z_half = fmax(zl, zr);

      // Account for riverwalls
      if (edge_flux_type[ki] == 1)
      {
        RiverWall_count = edge_river_wall_counter[ki];

        // Set central bed to riverwall elevation
        z_half = fmax(riverwall_elevation[RiverWall_count - 1], z_half);
      }

      // Define h left/right for Audusse flux method
      h_left = fmax(hle + zl - z_half, 0.);
      h_right = fmax(hre + zr - z_half, 0.);

      normal_x = normals[ki2];
      normal_y = normals[ki2 + 1];

      // Edge flux computation (triangle k, edge i)
      __flux_function_central(ql, qr,
                              h_left, h_right,
                              hle, hre,
                              normal_x, normal_y,
                              epsilon, z_half, limiting_threshold, g,
                              edgeflux, &max_speed_local, &pressure_flux,
                              hc, hc_n, low_froude);

      // Force weir discharge to match weir theory
      if (edge_flux_type[ki] == 1)
      {

        RiverWall_count = edge_river_wall_counter[ki];

        // printf("RiverWall_count %ld\n", RiverWall_count);

        ii = riverwall_rowIndex[RiverWall_count - 1] * ncol_riverwall_hydraulic_properties;

        // Get Qfactor index - multiply the idealised weir discharge by this constant factor
        // Get s1, submergence ratio at which we start blending with the shallow water solution
        // Get s2, submergence ratio at which we entirely use the shallow water solution
        // Get h1, tailwater head / weir height at which we start blending with the shallow water solution
        // Get h2, tailwater head / weir height at which we entirely use the shallow water solution
        Qfactor = riverwall_hydraulic_properties[ii];
        s1 = riverwall_hydraulic_properties[ii + 1];
        s2 = riverwall_hydraulic_properties[ii + 2];
        h1 = riverwall_hydraulic_properties[ii + 3];
        h2 = riverwall_hydraulic_properties[ii + 4];

        weir_height = fmax(riverwall_elevation[RiverWall_count - 1] - fmin(zl, zr), 0.); // Reference weir height

        // Use first-order h's for weir -- as the 'upstream/downstream' heads are
        //  measured away from the weir itself
        h_left_tmp = fmax(stage_centroid_values[k] - z_half, 0.);

        if (n >= 0)
        {
          h_right_tmp = fmax(stage_centroid_values[n] - z_half, 0.);
        }
        else
        {
          h_right_tmp = fmax(hc_n + zr - z_half, 0.);
        }

        // If the weir is not higher than both neighbouring cells, then
        // do not try to match the weir equation. If we do, it seems we
        // can get mass conservation issues (caused by large weir
        // fluxes in such situations)
        if (riverwall_elevation[RiverWall_count - 1] > fmax(zc, zc_n))
        {
          // Weir flux adjustment
          __adjust_edgeflux_with_weir(edgeflux, h_left_tmp, h_right_tmp, g,
                                      weir_height, Qfactor,
                                      s1, s2, h1, h2, &max_speed_local);
        }
      }

      // Multiply edgeflux by edgelength
      length = edgelengths[ki];
      edgeflux[0] = -edgeflux[0] * length;
      edgeflux[1] = -edgeflux[1] * length;
      edgeflux[2] = -edgeflux[2] * length;

      // bedslope_work contains all gravity related terms
      pressuregrad_work = length * (-g * 0.5 * (h_left * h_left - hle * hle - (hle + hc) * (zl - zc)) + pressure_flux);

      // Update timestep based on edge i and possibly neighbour n
      // NOTE: We should only change the timestep on the 'first substep'
      // of the timestepping method [substep_count==0]
      if (substep_count == 0)
      {

        // Compute the 'edge-timesteps' (useful for setting flux_update_frequency)
        edge_timestep = radii[k] * 1.0 / fmax(max_speed_local, epsilon);

        // Update the timestep
        if ((tri_full_flag[k] == 1))
        {
          if (max_speed_local > epsilon)
          {
            // Apply CFL condition for triangles joining this edge (triangle k and triangle n)

            // CFL for triangle k

            //local_timestep[0] = fmin(local_timestep[0], edge_timestep);
	          //atomicMin_double(local_timestep, edge_timestep);

            local_timestep = fmin(local_timestep, edge_timestep);

            speed_max_last = fmax(speed_max_last, max_speed_local);
          }
        }
      }

      local_stage_explicit_update = local_stage_explicit_update + edgeflux[0];
      local_xmom_explicit_update  = local_xmom_explicit_update + edgeflux[1];
      local_ymom_explicit_update  = local_ymom_explicit_update + edgeflux[2];

      // If this cell is not a ghost, and the neighbour is a
      // boundary condition OR a ghost cell, then add the flux to the
      // boundary_flux_integral
      if (((n < 0) & (tri_full_flag[k] == 1)) | ((n >= 0) && ((tri_full_flag[k] == 1) & (tri_full_flag[n] == 0))))
      {
        // boundary_flux_sum is an array with length = timestep_fluxcalls
        // For each sub-step, we put the boundary flux sum in.
        //boundary_flux_sum[substep_count] += edgeflux[0];
        local_boundary_flux_sum += edgeflux[0];
        
	      //atomicAdd((boundary_flux_sum+substep_count), edgeflux[0]);

        //printf(" k = %d  substep_count = %ld edge_flux %f bflux %f \n",k,substep_count, edgeflux[0], boundary_flux_sum[substep_count] );

        //printf('boundary_flux_sum_substep %e \n',boundary_flux_sum_substep);
        
        
      }

      local_xmom_explicit_update -= normals[ki2] * pressuregrad_work;
      local_ymom_explicit_update -= normals[ki2 + 1] * pressuregrad_work;

    } // End edge i (and neighbour n)

    // Keep track of maximal speeds
    if (substep_count == 0)
      max_speed[k] = speed_max_last; // max_speed;

    // Normalise triangle k by area and store for when all conserved
    // quantities get updated
    inv_area = 1.0 / areas[k];
    stage_explicit_update[k] = local_stage_explicit_update * inv_area;
    xmom_explicit_update[k]  = local_xmom_explicit_update * inv_area;
    ymom_explicit_update[k]  = local_ymom_explicit_update * inv_area;

    boundary_flux_sum_k_array[k] = local_boundary_flux_sum;
    timestep_k_array[k] = local_timestep;

  } // End triangle k


//  printf("cuda boundary_flux_sum_substep %f \n",boundary_flux_sum[substep_count]);
//  printf("cuda local_timestep            %f \n",local_timestep[0]);

}




// // Computational function for flux computation
// int main(int *argc, char*argv[])
// {
//   // local variables
//   long substep_count;
//   long number_of_elements =1024;
  
//   double limiting_threshold = 10 ;
//   long   low_froude;
//   double g;
//   double epsilon;

//   long ncol_riverwall_hydraulic_properties;
 
//   double local_timestep[1];      // InOut
//   double* boundary_flux_sum ;     // InOut
//   double* max_speed;             // InOut
//   double* stage_explicit_update; // InOut
//   double* xmom_explicit_update; // InOut
//   double* ymom_explicit_update ;// InOut

//   double* stage_centroid_values;
//   double* stage_edge_values;
//   double* xmom_edge_values ;
//   double* ymom_edge_values ;
//   double* bed_edge_values ;
//   double* height_edge_values ;
//   double* height_centroid_values;
//   double* bed_centroid_values ;
//   double* stage_boundary_values ;
//   double* xmom_boundary_values ;
//   double* ymom_boundary_values ;
//   double* areas ;
//   double* normals ;
//   double* edgelengths ;
//   double* radii ;
//   long* tri_full_flag ;
//   long* neighbours ;
//   long* neighbour_edges ;
//   long* edge_flux_type ;
//   long* edge_river_wall_counter ;
//   double* riverwall_elevation ;
//   long* riverwall_rowIndex ;
//   double* riverwall_hydraulic_properties;

//   unsigned int THREADS_PER_BLOCK;

//   long timestep_fluxcalls = 1;
//   long base_call = 1;
//   THREADS_PER_BLOCK = 256;
//   long NO_OF_BLOCKS = number_of_elements/THREADS_PER_BLOCK; 

//   __cuda_compute_fluxes_loop_1<<<NO_OF_BLOCKS,THREADS_PER_BLOCK>>>(local_timestep,        // InOut
//                                boundary_flux_sum,     // InOut
//                                max_speed,             // InOut
//                                stage_explicit_update, // InOut
//                                xmom_explicit_update,  // InOut
//                                ymom_explicit_update,  // InOut

//                                stage_centroid_values,
//                                stage_edge_values,
//                                xmom_edge_values,
//                                ymom_edge_values,
//                                bed_edge_values,
//                                height_edge_values,
//                                height_centroid_values,
//                                bed_centroid_values,
//                                stage_boundary_values,
//                                xmom_boundary_values,
//                                ymom_boundary_values,
//                                areas,
//                                normals,
//                                edgelengths,
//                                radii,
//                                tri_full_flag,
//                                neighbours,
//                                neighbour_edges,
//                                edge_flux_type,
//                                edge_river_wall_counter,
//                                riverwall_elevation,
//                                riverwall_rowIndex,
//                                riverwall_hydraulic_properties,

//                                number_of_elements,
//                                substep_count,
//                                ncol_riverwall_hydraulic_properties,
//                                epsilon,
//                                g,
//                                low_froude,
//                                limiting_threshold);

// }

